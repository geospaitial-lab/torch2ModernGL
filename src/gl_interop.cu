#include <stdint.h>

#ifdef _WIN32
#include <windows.h>
#endif
#include "utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

// from https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void deleter(void* device_pointer){
    gpuErrchk(hipFree(device_pointer));
}

torch::Tensor gl_texture_to_torch_cu(
    const int gl_object,
    const int width,
    const int height,
    const int components,
    const int dtype
){

    int bytes = 1;
    auto options = torch::TensorOptions().device(torch::kCUDA);
    if(dtype==0){
        bytes = 1;
        options = options.dtype(torch::kUInt8);
    } else if(dtype==1){
        bytes = 4;
        options = options.dtype(torch::kFloat32);
    } else if(dtype==2){
        bytes = 1;
        options = options.dtype(torch::kInt8);
    } else if(dtype==3){
        bytes = 2;
        options = options.dtype(torch::kInt16);
    } else if(dtype==4){
        bytes = 4;
        options = options.dtype(torch::kInt32);
    } else {
    printf("Invalid dtype!");
    }

    struct hipGraphicsResource *cuda_texture;
    gpuErrchk(hipGraphicsGLRegisterImage(&cuda_texture, gl_object, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly));
    gpuErrchk(hipGraphicsMapResources(1, &cuda_texture, 0));

    hipArray_t cuda_texture_device_array;
    gpuErrchk(hipGraphicsSubResourceGetMappedArray(&cuda_texture_device_array, cuda_texture, 0, 0));

	void* devicePtr = nullptr;
	gpuErrchk(hipMalloc(&devicePtr, width * height * components * bytes));

	gpuErrchk(hipMemcpy2DFromArray(devicePtr, width * components * bytes, cuda_texture_device_array
	    , 0, 0, width * components * bytes, height, hipMemcpyDeviceToDevice));

	auto out_tensor = torch::from_blob(devicePtr, {height, width, components},
	    deleter, options);

	devicePtr = nullptr;

    gpuErrchk(hipGraphicsUnmapResources(1, &cuda_texture, 0));

    gpuErrchk(hipGraphicsUnregisterResource(cuda_texture));

    return out_tensor;
}


void torch_to_gl_texture_cu(
    const torch::Tensor tensor,
    const int gl_object,
    const int width,
    const int height,
    const int element_bytes
){
    struct hipGraphicsResource *cuda_texture;
    gpuErrchk(hipGraphicsGLRegisterImage(&cuda_texture, gl_object, GL_TEXTURE_2D,
        hipGraphicsRegisterFlagsWriteDiscard));
    gpuErrchk(hipGraphicsMapResources(1, &cuda_texture, 0));

    hipArray_t cuda_texture_device_array;
    gpuErrchk(hipGraphicsSubResourceGetMappedArray(&cuda_texture_device_array, cuda_texture, 0, 0));

	gpuErrchk(hipMemcpy2DToArray(cuda_texture_device_array, 0, 0, tensor.data_ptr(), width * element_bytes,
	    width * element_bytes, height, hipMemcpyDeviceToDevice));

    gpuErrchk(hipGraphicsUnmapResources(1, &cuda_texture, 0));

    gpuErrchk(hipGraphicsUnregisterResource(cuda_texture));
}